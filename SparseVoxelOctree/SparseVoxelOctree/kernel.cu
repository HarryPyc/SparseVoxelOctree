
#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include "Mesh.h"
#include "Voxel.cuh"

int main() {
	/*if (hipSetDevice(0) != hipSuccess) {
		printf("hipSetDevice Failed");
		return 0; 
	}*/
	CudaMesh cuMesh;
	Mesh mesh("asset/model/bunny.obj");
	mesh.UploatToDevice(cuMesh);
	
	return 0;
}
