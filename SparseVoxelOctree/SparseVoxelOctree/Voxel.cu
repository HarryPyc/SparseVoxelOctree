#include "hip/hip_runtime.h"
#include "Voxel.cuh"
#include "Mesh.h"
#include ""
texture<float4, 2, hipReadModeElementType> frontTex, backTex;

typedef unsigned int uint;
__device__ __host__ glm::vec4 ConvUintToVec4(unsigned int val)
{
	glm::vec4 res(float((val & 0x000000FF)), float((val & 0x0000FF00) >> 8U), float((val & 0x00FF0000) >> 16U), float((val & 0xFF000000) >> 24U));
	return res / 255.f;
}
__device__ __host__ unsigned int ConvVec4ToUint(glm::vec4 val) {
	val *= 255.f;
	return (uint(val.w) & 0x000000FF) << 24U | (uint(val.z) & 0x000000FF) << 16U | (uint(val.y) & 0x000000FF) << 8U | (uint(val.x) & 0x000000FF);
}

__host__  __device__ Voxel::Voxel()
{
	c = 0, n = 0;
}

__host__  __device__ Voxel::~Voxel() {

}

__host__ __device__ void Voxel::GetInfo(glm::vec3& color, glm::vec3& normal) {
	color = glm::vec3(float((c & 0x000000FF)), float((c & 0x0000FF00) >> 8U), float((c & 0x00FF0000) >> 16U));
	normal = glm::vec3(float((n & 0x000000FF)), float((n & 0x0000FF00) >> 8U), float((n & 0x00FF0000) >> 16U));
	color /= 255.f;
	normal = (normal / 255.f) * 2.f - 1.f;
}

__host__ __device__ void Voxel::SetInfo(glm::vec3 color, glm::vec3 normal) {
	color *= 255.f;
	c = ((uint(color.z) & 0x000000FF) << 16U | (uint(color.y) & 0x000000FF) << 8U | (uint(color.x) & 0x000000FF));
	normal = (normal + 1.f) / 2.f * 255.f;
	n = ((uint(normal.z) & 0x000000FF) << 16U | (uint(normal.y) & 0x000000FF) << 8U | (uint(normal.x) & 0x000000FF));
}

//transfer 3D index to 1D array index
__device__ inline size_t ToArrayIdx(glm::uvec3 coord) {
	return coord.z + coord.y * voxelDim + coord.x * voxelDim * voxelDim;
}
//Get voxel index for current position
__device__ inline glm::uvec3 GetVoxelIndex(glm::vec3 pos, glm::vec3 minAABB, float delta) {
	return glm::min(glm::uvec3((pos - minAABB)/delta * float(voxelDim)), glm::uvec3(voxelDim - 1));
}
//Get world position given voxel index
__device__ inline glm::vec3 GetVoxelWorldPos(glm::uvec3 idx, glm::vec3 minAABB, float delta) {
	return glm::vec3(idx) * delta / float(voxelDim) + minAABB;
}

__device__ inline glm::vec3 WorldSpaceInterpolation(glm::vec3 A, glm::vec3 B, glm::vec3 C, glm::vec3 P) {
	glm::vec3 AB = B - A, AC = C - A, AP = P - A, N = glm::cross(AB, AC);
	float DotNN = glm::dot(N, N);
	glm::vec3 uvw;
	uvw[1] = glm::dot(glm::cross(AP, AC), N) / DotNN;
	uvw[2] = glm::dot(glm::cross(AB, AP), N) / DotNN;
	uvw[0] = 1.f - uvw[1] - uvw[2];
	return uvw;
}

__device__ inline bool VoxelTriangleIntersection(Triangle tri, glm::vec3 vMinAABB) {
	const float dotnp = glm::dot(tri.n, vMinAABB);
	if ((dotnp + tri.d1) * (dotnp + tri.d2) > 0)
		return false;
	bool xy, xz, yz;
	xy = (glm::dot(tri.ne_xy[0], glm::vec2(vMinAABB.x, vMinAABB.y)) + tri.de_xy[0]) >= 0 &&
		(glm::dot(tri.ne_xy[1], glm::vec2(vMinAABB.x, vMinAABB.y)) + tri.de_xy[1]) >= 0 &&
		(glm::dot(tri.ne_xy[2], glm::vec2(vMinAABB.x, vMinAABB.y)) + tri.de_xy[2]) >= 0;
	xz = (glm::dot(tri.ne_xz[0], glm::vec2(vMinAABB.x, vMinAABB.z)) + tri.de_xz[0]) >= 0 &&
		(glm::dot(tri.ne_xz[1], glm::vec2(vMinAABB.x, vMinAABB.z)) + tri.de_xz[1]) >= 0 &&
		(glm::dot(tri.ne_xz[2], glm::vec2(vMinAABB.x, vMinAABB.z)) + tri.de_xz[2]) >= 0;
	yz = (glm::dot(tri.ne_yz[0], glm::vec2(vMinAABB.y, vMinAABB.z)) + tri.de_yz[0]) >= 0 &&
		(glm::dot(tri.ne_yz[1], glm::vec2(vMinAABB.y, vMinAABB.z)) + tri.de_yz[1]) >= 0 &&
		(glm::dot(tri.ne_yz[2], glm::vec2(vMinAABB.y, vMinAABB.z)) + tri.de_yz[2]) >= 0;
	return xy || xz || yz;
}

__global__ void VoxelizationKernel(Voxel* voxelList, CudaMesh mesh, const unsigned short voxelDim) {
#ifdef TRIANGLE_PER_THREAD
	const unsigned idx = blockDim.x * blockIdx.x + threadIdx.x;
	if (idx >= mesh.triNum) return;
	const Triangle tri = mesh.d_tri[idx];
	const glm::vec3 v0(mesh.d_v[3 * tri.i0], mesh.d_v[3 * tri.i0 + 1], mesh.d_v[3 * tri.i0 + 2]),
					v1(mesh.d_v[3 * tri.i1], mesh.d_v[3 * tri.i1 + 1], mesh.d_v[3 * tri.i1 + 2]),
					v2(mesh.d_v[3 * tri.i2], mesh.d_v[3 * tri.i2 + 1], mesh.d_v[3 * tri.i2 + 2]),
					n0(mesh.d_n[3 * tri.i0], mesh.d_n[3 * tri.i0 + 1], mesh.d_n[3 * tri.i0 + 2]),
					n1(mesh.d_n[3 * tri.i1], mesh.d_n[3 * tri.i1 + 1], mesh.d_n[3 * tri.i1 + 2]),
					n2(mesh.d_n[3 * tri.i2], mesh.d_n[3 * tri.i2 + 1], mesh.d_n[3 * tri.i2 + 2]);
	const float vDelta = mesh.delta / float(voxelDim);

	glm::vec3 maxAABB(glm::max(v0, glm::max(v1, v2))), minAABB(glm::min(v0, glm::min(v1, v2)));
	glm::uvec3 minVoxel = GetVoxelIndex(minAABB, mesh.minAABB, mesh.delta),
			   maxVoxel = GetVoxelIndex(maxAABB, mesh.minAABB, mesh.delta);
	//printf("maxVoxel:(%i, %i, %i)\n", maxVoxel.x, maxVoxel.y, maxVoxel.z);
	for(uint i = minVoxel.x; i <= maxVoxel.x; i++)
		for (uint j = minVoxel.y; j <= maxVoxel.y; j++)
			for (uint k = minVoxel.z; k <= maxVoxel.z; k++) {
				glm::vec3 voxelPos = GetVoxelWorldPos(glm::uvec3(i, j, k), mesh.minAABB, mesh.delta);
				if (VoxelTriangleIntersection(tri, voxelPos - vDelta/2.f)) {
					glm::vec3 uvw = WorldSpaceInterpolation(v0, v1, v2, voxelPos);
					Voxel voxel;
					glm::vec3 color(1, 1, 0), normal = glm::normalize(uvw[0] * n0 + uvw[1] * n1 + uvw[2] * n2);
					voxel.SetInfo(color, normal);
					voxelList[ToArrayIdx(glm::uvec3(i, j, k))] = voxel;
				}

			}

#endif // TRIANGLE_PER_THREAD
}

__global__ void PreProcessTriangleKernel(CudaMesh mesh, const unsigned short voxelDim) {
	const unsigned idx = blockDim.x * blockIdx.x + threadIdx.x;
	if (idx >= mesh.triNum) return;
	Triangle tri;
	tri.i0 = mesh.d_idx[3 * idx], tri.i1 = mesh.d_idx[3 * idx + 1], tri.i2 = mesh.d_idx[3 * idx + 2];
	glm::vec3 v[3];
	v[0] = glm::vec3(mesh.d_v[3 * tri.i0], mesh.d_v[3 * tri.i0 + 1], mesh.d_v[3 * tri.i0 + 2]);
	v[1] = glm::vec3(mesh.d_v[3 * tri.i1], mesh.d_v[3 * tri.i1 + 1], mesh.d_v[3 * tri.i1 + 2]);
	v[2] = glm::vec3(mesh.d_v[3 * tri.i2], mesh.d_v[3 * tri.i2 + 1], mesh.d_v[3 * tri.i2 + 2]);
	const glm::vec3 e[3] = { v[1] - v[0], v[2] - v[1], v[0] - v[2] };
	const float delta = mesh.delta / (float)voxelDim;
	//Pre-compute parameters for voxel triangle intersection
	tri.n = glm::cross(e[0], e[1]);
	glm::vec3 c(tri.n.x > 0 ? delta : 0, tri.n.y > 0 ? delta : 0, tri.n.z > 0 ? delta : 0);
	tri.d1 = glm::dot(tri.n, c - v[0]), tri.d2 = glm::dot(tri.n, delta - c - v[0]);

	for (int i = 0; i < 3; i++) {
		tri.ne_xy[i] = glm::vec2(-e[i].y, e[i].x) * (tri.n.z >= 0.f ? 1.f : -1.f);
		tri.ne_xz[i] = glm::vec2(-e[i].z, e[i].x) * (tri.n.y >= 0.f ? 1.f : -1.f);
		tri.ne_yz[i] = glm::vec2(-e[i].z, e[i].y) * (tri.n.x >= 0.f ? 1.f : -1.f);

		tri.de_xy[i] = -glm::dot(tri.ne_xy[i], glm::vec2(v[i].x, v[i].y)) + glm::max(0.f, delta * tri.ne_xy[i].x)
			+ glm::max(0.f, delta * tri.ne_xy[i].y);
		tri.de_xz[i] = -glm::dot(tri.ne_xz[i], glm::vec2(v[i].x, v[i].z)) + glm::max(0.f, delta * tri.ne_xz[i].x)
			+ glm::max(0.f, delta * tri.ne_xz[i].y);
		tri.de_yz[i] = -glm::dot(tri.ne_yz[i], glm::vec2(v[i].y, v[i].z)) + glm::max(0.f, delta * tri.ne_yz[i].x)
			+ glm::max(0.f, delta * tri.ne_yz[i].y);
	}
	//Write to global memory
	mesh.d_tri[idx] = tri;

}

__global__ void RayMarchingKernel(unsigned int* d_pbo, Voxel* voxelList, CudaMesh mesh, const unsigned short voxelDim, const unsigned int w, const unsigned int h) {
	const unsigned int x = blockDim.x * blockIdx.x + threadIdx.x,
		y = blockDim.y * blockIdx.y + threadIdx.y;
	if (x >= w || y >= h) return;
	const float u = float(x) / float(w), v = float(y) / float(h);
	float4 frontSample = tex2D(frontTex, u, v), backSample = tex2D(backTex, u, v);
	if (frontSample.w < 1.f) return;

	glm::vec3 frontPos(frontSample.x, frontSample.y, frontSample.z),
		backPos(backSample.x, backSample.y, backSample.z);
	glm::vec3 dir = backPos - frontPos;
	const float stepSize = mesh.delta / voxelDim , dirLength = glm::length(dir);
	const unsigned maxStep = dirLength / stepSize;
	dir /= dirLength;//Normalize
	glm::vec3 curPos = frontPos;
	glm::uvec3 voxelPos;
	//Trace voxels
	for (int i = 0; i < maxStep; i++) {
		curPos +=  dir * stepSize;
		voxelPos = GetVoxelIndex(curPos, mesh.minAABB, mesh.delta);

		Voxel voxel = voxelList[ToArrayIdx(voxelPos)];
		if (voxel.n != 0) {
			
			d_pbo[y * w + x] = 0x0000FFFF;
			break;
		}
	}

}

void Voxelization(CudaMesh& cuMesh, Voxel*& d_voxel)
{
	hipError_t cudaStatus;
	//PreProcess Triangle
	cudaStatus = hipMalloc((void**)&cuMesh.d_tri, cuMesh.triNum * sizeof(Triangle));
	if (cudaStatus != hipSuccess) printf("d_tri hipMalloc Failed\n");
	dim3 blockDim = 256, gridDim = cuMesh.triNum / blockDim.x + 1;
	PreProcessTriangleKernel <<< gridDim, blockDim >>> (cuMesh, voxelDim);
	cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess) printf("PreprocessTriangle Launch Kernel Failed\n");
	cudaStatus = hipDeviceSynchronize();
	if (cudaStatus != hipSuccess) printf("hipDeviceSynchronize Failed\n");
	cudaStatus = hipFree(cuMesh.d_idx);
	if (cudaStatus != hipSuccess) printf("d_idx hipFree Failed, error: %s\n", hipGetErrorString(cudaStatus));

	size_t voxelSize = voxelDim * voxelDim * voxelDim * sizeof(Voxel);

	cudaStatus = hipMalloc((void**)&d_voxel, voxelSize);
	if (cudaStatus != hipSuccess) printf("d_voxel hipMalloc Failed\n");
	
#ifndef TRIANGLE_PER_THREAD
	blockDim = dim3(8, 8, 8), gridDim = dim3(voxelDim / blockDim.x, voxelDim / blockDim.y, voxelDim / blockDim.z);
#endif // TRIANGLE_PER_THREAD	
	VoxelizationKernel << <gridDim, blockDim >> > (d_voxel, cuMesh, voxelDim);
	cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess) printf("cuda Launch Kernel Failed\n");

	cudaStatus = hipDeviceSynchronize();
	if (cudaStatus != hipSuccess) printf("hipDeviceSynchronize Failed\n");
	//Free CudaMesh
	cudaStatus = hipFree(cuMesh.d_v);
	if (cudaStatus != hipSuccess) printf("d_v hipFree Failed, error: %s\n", hipGetErrorString(cudaStatus));
	cudaStatus = hipFree(cuMesh.d_n);
	if (cudaStatus != hipSuccess) printf("d_n hipFree Failed, error: %s\n", hipGetErrorString(cudaStatus));
	cudaStatus = hipFree(cuMesh.d_tri);
	if (cudaStatus != hipSuccess) printf("d_tri hipFree Failed, error: %s\n", hipGetErrorString(cudaStatus));

	printf("voxelization finished\n");
}

void RunRayMarchingKernel(unsigned int* d_pbo, hipArray_t front, hipArray_t back, Voxel* d_voxel, CudaMesh cuMesh, const unsigned w, const unsigned h)
{
	hipChannelFormatDesc format = hipCreateChannelDesc(32, 32, 32, 32, hipChannelFormatKindFloat);
	hipError_t cudaStatus;
	if (hipBindTextureToArray(&frontTex, front, &format) != hipSuccess)
		printf("front texture bind failed\n");
	if (hipBindTextureToArray(&backTex, back, &format) != hipSuccess)
		printf("back texture bind failed\n");
	//launch cuda kernel
	dim3 blockDim(16, 16, 1), gridDim(w / blockDim.x + 1, h / blockDim.y + 1, 1);
	RayMarchingKernel << <gridDim, blockDim >> > (d_pbo, d_voxel, cuMesh, voxelDim, w, h);
	cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess) printf("raymarching cuda Launch Kernel Failed\n");
	cudaStatus = hipDeviceSynchronize();
	if(cudaStatus != hipSuccess)
		printf("hipDeviceSynchronize Failed, error: %s\n", hipGetErrorString(cudaStatus));

	if (hipUnbindTexture(frontTex) != hipSuccess)
		printf("cuda unbind texture failed\n");
	if (hipUnbindTexture(backTex) != hipSuccess)
		printf("cuda unbind texture failed\n");
}

void initCudaTexture()
{
	frontTex.addressMode[0] = hipAddressModeWrap;
	frontTex.addressMode[1] = hipAddressModeWrap;
	frontTex.filterMode = hipFilterModeLinear;
	frontTex.normalized = true;

	backTex.addressMode[0] = hipAddressModeWrap;
	backTex.addressMode[1] = hipAddressModeWrap;
	backTex.filterMode = hipFilterModeLinear;
	backTex.normalized = true;
}
