#include "hip/hip_runtime.h"
#include "Octree.cuh"
#include ""
#include <glm/gtc/integer.hpp>
#include <time.h>
#include "Morton.cuh"
#define STB_IMAGE_IMPLEMENTATION
#include <stb_image.h>
#define NULLPTR 0U
#define MARKED 0xFFFFFFFF
extern VoxelizationInfo Info;
extern uint h_MIPMAP;
__constant__ VoxelizationInfo d_Info;
__constant__ int curDepth, MIPMAP, d_MAX_DEPTH;
Node root;
__device__ int nodeCounter, dynamicNodeCounter, traverseCounter;

texture<float4, 2, hipReadModeElementType> backTex;
texture<uint4, 1, hipReadModeElementType> octree;
texture<float4, hipTextureTypeCubemap, hipReadModeElementType> skyBox;

__host__ __device__ Node::Node() : voxel() {
	ptr = NULLPTR;
}

__host__ __device__ Node::Node(uint _ptr, Voxel _vox) {
	ptr = _ptr;
	voxel = _vox;
}
__host__ __device__ Node::~Node() {
}

void initCudaTexture()
{
	backTex.addressMode[0] = hipAddressModeWrap;
	backTex.addressMode[1] = hipAddressModeWrap;
	backTex.filterMode = hipFilterModeLinear;
	backTex.normalized = true;

	octree.addressMode[0] = hipAddressModeWrap;
	octree.filterMode = hipFilterModePoint;
	octree.normalized = false;
}

__global__ void OctreeConstructKernel(Node* d_node, Voxel* d_voxel, Voxel* d_nextVoxel, int* d_ptr, int* d_nextPtr) {
	//__shared__ Voxel voxels[8 * 256];
	const int idx = blockDim.x * blockIdx.x + threadIdx.x;
	if (idx >= 1 << 3 * (curDepth - 1)) return;

	//for (int i = 0; i < 8; i++)
	//	voxels[threadIdx.x * 8 + i] = d_voxel[idx * 8 + i];//copy to shared memory

	glm::vec3 color(0.f), normal(0.f);
	int counter = 0;
	int bitMask = 0;
	Voxel v;
	for (int i = 0; i < 8; i++) {
		//v = voxels[threadIdx.x * 8 + i];
		v = d_voxel[idx * 8 + i];
		if (!v.empty()) {
			glm::vec3 c, n;
			v.GetInfo(c, n);
			color += c, normal += n;
			counter++;
			bitMask |= 1 << i;
		}
	}
	if (counter > 0) {
		size_t arrayIdx = atomicAdd(&nodeCounter, 8);
		for (int i = 0; i < 8; i++) {
			int ptr = (1 << curDepth) == d_Info.Dim ? NULLPTR : d_ptr[idx * 8 + i];
			d_node[arrayIdx + i] = Node(ptr, d_voxel[idx * 8 + i]);
		}
		Voxel voxel;
		voxel.SetInfo(color / float(counter), glm::normalize(normal / float(counter)));
		voxel.n |= bitMask << 24U;
		d_nextVoxel[idx] = voxel;
		d_nextPtr[idx] = arrayIdx;
	}

}

void OctreeConstruction(Node*& d_node, Voxel*& d_voxel)
{
	clock_t t = clock();
	const int MAX_DEPTH = log2(Info.Dim);
	gpuErrchk(hipMemcpyToSymbol(HIP_SYMBOL(d_MAX_DEPTH), &MAX_DEPTH, sizeof(int)));
	//size_t NODE_SIZE= ((1 << 3 * (MAX_DEPTH + 1)) - 1) / 7 * sizeof(Node);
	size_t NODE_SIZE = 5e6 * sizeof(Node);

	gpuErrchk(hipMalloc((void**)&d_node, NODE_SIZE));
	gpuErrchk(hipMemset(d_node, 0, NODE_SIZE));
	Voxel* d_nextVoxel; int* d_ptr, *d_nextPtr; //lower level voxel
	int h_nodeCounter = 0;
	gpuErrchk(hipMemcpyToSymbol(HIP_SYMBOL(nodeCounter), &h_nodeCounter, sizeof(int)));
	gpuErrchk(hipMemcpyToSymbol(HIP_SYMBOL(d_Info), &Info, sizeof(VoxelizationInfo)));

	for (int i = MAX_DEPTH; i > 0; i--) {
		gpuErrchk(hipMemcpyToSymbol(HIP_SYMBOL(curDepth), &i, sizeof(int)));
		size_t nextSize = 1 << 3 * (i - 1);
		gpuErrchk(hipMalloc((void**)&d_nextVoxel, nextSize * sizeof(Voxel)));
		gpuErrchk(hipMemset(d_nextVoxel, 0, nextSize * sizeof(Voxel)));
		gpuErrchk(hipMalloc((void**)&d_nextPtr, nextSize * sizeof(int)));
		gpuErrchk(hipMemset(d_nextPtr, 0, nextSize * sizeof(int)));

		dim3 blockDim = 256, gridDim = (1 << 3 * i) / 8 / blockDim.x + 1;
		OctreeConstructKernel << <gridDim, blockDim>> > (d_node, d_voxel, d_nextVoxel, d_ptr, d_nextPtr);
		gpuErrchk(hipGetLastError());
		gpuErrchk(hipDeviceSynchronize());

		gpuErrchk(hipMemcpyFromSymbol(&h_nodeCounter, HIP_SYMBOL(nodeCounter), sizeof(int)));
		gpuErrchk(hipFree(d_voxel));
		if(i != MAX_DEPTH)
			gpuErrchk(hipFree(d_ptr));
		d_voxel = d_nextVoxel;
		d_ptr = d_nextPtr;
	}

	gpuErrchk(hipMemcpy(&root.ptr, d_ptr, sizeof(int), hipMemcpyDeviceToHost));
	gpuErrchk(hipMemcpy(&root.voxel, d_voxel, sizeof(Voxel), hipMemcpyDeviceToHost));

	gpuErrchk(hipFree(d_voxel));
	gpuErrchk(hipFree(d_ptr));
	gpuErrchk(hipMemcpyFromSymbol(&h_nodeCounter, HIP_SYMBOL(nodeCounter), sizeof(int)));
	t = clock() - t;
	printf("Octree Construction Complete, %i total nodes in %f sec\n", h_nodeCounter, (float)t / CLOCKS_PER_SEC);
}

__device__ inline int hasNode(Node* d_node, Node root, int& arrayIdx) {
	int offset = arrayIdx;
	int rootPos, i;
	for (i = 0; i < curDepth - 1; i++) {
		int s = 1 << 3 * (curDepth - i - 1);//how many leaf nodes each node has.
		offset = arrayIdx / s;
		if (root.ptr != NULLPTR && root.hasVoxel(offset)) {
			rootPos = root.ptr + offset;
			root = d_node[root.ptr + offset];
		}
		else
			break;
		arrayIdx -= offset * s;
	}
	arrayIdx = rootPos;
	return i;
}
__global__ void OctreeUpdateKernel(Node* d_node, Node root, Voxel* d_voxel, Voxel* d_nextVoxel, int* d_ptr, int* d_nextPtr) {
	const int idx = blockDim.x * blockIdx.x + threadIdx.x;
	if (idx >= 1 << 3 * (curDepth - 1)) return;

	glm::vec3 color(0.f), normal(0.f);
	int counter = 0;
	int bitMask = 0;
	Voxel v;
	int staticParLevel, staticParPos = idx * 8;
	staticParLevel = hasNode(d_node, root, staticParPos);
	for (int i = 0; i < 8; i++) {
		v = d_voxel[idx * 8 + i];
		if (!v.empty()) {
			glm::vec3 c, n;
			v.GetInfo(c, n);
			color += c, normal += n;
			counter++;
			bitMask |= 1 << i;
		}
	}
	if (counter > 0) {
		if (staticParLevel == curDepth - 1 && d_node[staticParPos].ptr != NULLPTR) {
			Node sRoot = d_node[staticParPos];
			for (int i = 0; i < 8; i++) {
				if (d_node[sRoot.ptr + i].voxel.empty()) {
					int ptr = (1 << curDepth) == d_Info.Dim ? NULLPTR : d_ptr[idx * 8 + i];
					d_node[sRoot.ptr + i] = Node(ptr, d_voxel[idx * 8 + i]);
				}
			}
			sRoot.voxel.n |= bitMask << 24U;
			d_node[staticParPos] = sRoot;
		}
		else {
			size_t arrayIdx = atomicAdd(&dynamicNodeCounter, 8) + nodeCounter;
			for (int i = 0; i < 8; i++) {
				int ptr = (1 << curDepth) == d_Info.Dim ? NULLPTR : d_ptr[idx * 8 + i];
				d_node[arrayIdx + i] = Node(ptr, d_voxel[idx * 8 + i]);
			}
			Voxel voxel;
			voxel.SetInfo(color / float(counter), glm::normalize(normal / float(counter)));
			voxel.n |= bitMask << 24U;
			d_nextVoxel[idx] = voxel;
			d_nextPtr[idx] = arrayIdx;
		}
	}
}

void OctreeUpdate(Node*& d_node, Voxel*& d_voxel)
{
	clock_t t = clock();
	const int MAX_DEPTH = log2(Info.Dim);

	Voxel* d_nextVoxel; int* d_ptr, * d_nextPtr; //lower level voxel
	int h_nodeCounter = 0;
	gpuErrchk(hipMemcpyToSymbol(HIP_SYMBOL(d_Info), &Info, sizeof(VoxelizationInfo)));
	gpuErrchk(hipMemcpyToSymbol(HIP_SYMBOL(dynamicNodeCounter), &h_nodeCounter, sizeof(int)));

	for (int i = MAX_DEPTH; i > 0; i--) {
		gpuErrchk(hipMemcpyToSymbol(HIP_SYMBOL(curDepth), &i, sizeof(int)));
		size_t nextSize = 1 << 3 * (i - 1);
		gpuErrchk(hipMalloc((void**)&d_nextVoxel, nextSize * sizeof(Voxel)));
		gpuErrchk(hipMemset(d_nextVoxel, 0, nextSize * sizeof(Voxel)));
		gpuErrchk(hipMalloc((void**)&d_nextPtr, nextSize * sizeof(int)));
		gpuErrchk(hipMemset(d_nextPtr, 0, nextSize * sizeof(int)));

		dim3 blockDim = 256, gridDim = (1 << 3 * i) / 8 / blockDim.x + 1;
		OctreeUpdateKernel << <gridDim, blockDim >> > (d_node, root, d_voxel, d_nextVoxel, d_ptr, d_nextPtr);
		gpuErrchk(hipGetLastError());
		gpuErrchk(hipDeviceSynchronize());

		gpuErrchk(hipMemcpyFromSymbol(&h_nodeCounter, HIP_SYMBOL(dynamicNodeCounter), sizeof(int)));
		gpuErrchk(hipFree(d_voxel));
		if (i != MAX_DEPTH)
			gpuErrchk(hipFree(d_ptr));
		d_voxel = d_nextVoxel;
		d_ptr = d_nextPtr;
	}

	//gpuErrchk(hipMemcpy(&root.ptr, d_ptr, sizeof(int), hipMemcpyDeviceToHost));
	//gpuErrchk(hipMemcpy(&root.voxel, d_voxel, sizeof(Voxel), hipMemcpyDeviceToHost));

	gpuErrchk(hipFree(d_voxel));
	gpuErrchk(hipFree(d_ptr));
	gpuErrchk(hipMemcpyFromSymbol(&h_nodeCounter, HIP_SYMBOL(dynamicNodeCounter), sizeof(int)));

	t = clock() - t;
	printf("Octree Update Complete, %i total nodes in %f sec\n", h_nodeCounter, (float)t / CLOCKS_PER_SEC);
}



struct Ray {
	glm::vec3 o, d, invD;
	uint depth; bool inside;
	__device__ Ray(glm::vec3 origin, glm::vec3 dir, uint Depth = 0, bool Inside = false) 
		: o(origin), d(dir), depth(Depth), inside(Inside) {
		invD = glm::vec3(1.f) / dir;
	}
	__device__ ~Ray() {};
	__device__ inline bool RayAABBIntersection(glm::vec3 minAABB, glm::vec3 maxAABB, float &t) {
		glm::vec3 t0s = (minAABB - o) * invD;
		glm::vec3 t1s = (maxAABB - o) * invD;

		glm::vec3 tsmaller = glm::min(t0s, t1s);
		glm::vec3 tbigger = glm::max(t0s, t1s);

		float tmin = glm::max(-999.f, glm::max(tsmaller[0], glm::max(tsmaller[1], tsmaller[2])));
		float tmax = glm::min(999.f, glm::min(tbigger[0], glm::min(tbigger[1], tbigger[2])));
		t = (tmin + tmax) / 2.f;
		return (tmin < tmax) && tmax > 0.f;
	}
};
struct HitInfo {
	glm::uvec3 idx;
	float t;
	__device__ HitInfo() {};
	__device__ HitInfo(glm::uvec3 _idx, float _t) : idx(_idx), t(_t) {};
};

__device__ Voxel OctreeTraverse(Node* d_node, Node root, Ray ray, glm::vec3 minAABB, uint currentLevel, float& t) {
	//atomicAdd(&traverseCounter, 1U);
	if (currentLevel == MIPMAP)
		return root.voxel;
	if (root.ptr == NULLPTR)
		return Voxel();
	
	currentLevel++;
	HitInfo hits[8];
	int counter = 0;
	float temp = 999.f;

	const float delta = d_Info.delta / float((1 << (currentLevel)));
	for (int i = 0; i < 8; i++) {
		glm::uvec3 idx(i & 1, (i >> 1) & 1, i >> 2);
		glm::vec3 _minAABB = minAABB + glm::vec3(idx) * delta;
		float _t;
		if (root.hasVoxel(i) && ray.RayAABBIntersection(_minAABB, _minAABB + delta, _t)) {// !d_node[root.ptr + i].voxel.empty()
			HitInfo hit(idx, _t);
			hits[counter++] = hit;

		}
	}
	for(int i = 0; i < counter - 1; i++)
		for (int j = 0; j < counter - i - 1; j++) {
			if (hits[j].t > hits[j + 1].t) {
				HitInfo temp = hits[j];
				hits[j] = hits[j + 1];
				hits[j + 1] = temp;
			}
		}
	for (int i = 0; i < counter; i++) {
		Node _root = d_node[root.ptr + hits[i].idx.x + hits[i].idx.y * 2 + hits[i].idx.z * 4];
		glm::vec3 _minAABB = minAABB + glm::vec3(hits[i].idx) * delta;
		Voxel voxel = OctreeTraverse(d_node, _root, ray, _minAABB, currentLevel,  t);
		if (!voxel.empty()) {
			glm::vec3 c;
			glm::vec3 n;
			voxel.GetInfo(c, n);

			if (currentLevel == MIPMAP)
				t = hits[i].t;
			return voxel;
		}
	}
	return Voxel();
}

__global__ void RayCastKernel(uint* d_pbo, Node* d_node, const uint w, const uint h, Node root) {
	const unsigned int x = blockDim.x * blockIdx.x + threadIdx.x,
			y = blockDim.y * blockIdx.y + threadIdx.y;
	if (x >= w || y >= h) return;
	d_pbo[y * w + x] = 0;
	const float u = float(x) / float(w), v = float(y) / float(h);
	float4 backSample = tex2D(backTex, u, v);
		
	glm::vec3 dir = glm::normalize(glm::vec3(backSample.x, backSample.y, backSample.z));
	Ray ray(d_Info.camPos, dir, 0);

	glm::vec3 color(0.f);
	float t;
	if (ray.RayAABBIntersection(d_Info.minAABB, d_Info.minAABB + d_Info.delta, t)) {
		t = 999.f;
		Voxel voxel = OctreeTraverse(d_node, root, ray, d_Info.minAABB, 0, t);
		glm::vec3 pos = ray.o + t * ray.d;
		if(!voxel.empty())
			color = voxel.PhongLighting(pos);
		else {
			float4 texel = texCubemap(skyBox, dir.x, dir.y, dir.z);
			color = glm::vec4(texel.x, texel.y, texel.z, 1.f);
		}

	}
	else {
		float4 texel = texCubemap(skyBox, dir.x, dir.y, dir.z);
		color = glm::vec3(texel.x, texel.y, texel.z);
	}
	//Gamma Correction
	color = glm::pow(color, glm::vec3(1.f / 2.2f));
	d_pbo[y * w + x] = ConvVec4ToUint(glm::vec4(color, 1.f));

}


void RayCastingOctree(uint* d_pbo, glm::vec3 h_camPos, hipArray_t back, Node* d_node)
{
	gpuErrchk(hipMemcpyToSymbol(HIP_SYMBOL(d_Info), &Info, sizeof(VoxelizationInfo)));

	hipMemcpyToSymbol(HIP_SYMBOL(MIPMAP), &h_MIPMAP, 4);
	uint h_tCounter = 0;
	hipMemcpyToSymbol(HIP_SYMBOL(traverseCounter), &h_tCounter, 4);

	hipChannelFormatDesc format = hipCreateChannelDesc(32, 32, 32, 32, hipChannelFormatKindFloat);

	gpuErrchk(hipBindTextureToArray(&backTex, back, &format));

	//launch cuda kernel
	dim3 blockDim(16, 16, 1), gridDim(WINDOW_WIDTH / blockDim.x + 1, WINDOW_HEIGHT / blockDim.y + 1, 1);
	RayCastKernel << <gridDim, blockDim >> > (d_pbo, d_node, WINDOW_WIDTH, WINDOW_HEIGHT, root);
	gpuErrchk(hipGetLastError());
	gpuErrchk(hipDeviceSynchronize());

	hipMemcpyFromSymbol(&h_tCounter, HIP_SYMBOL(traverseCounter), 4);
	//printf("Traverse Count: %u\n", h_tCounter);
	gpuErrchk(hipUnbindTexture(backTex));
}

void initSkyBox() {
	std::string faces[6]{
		"asset/texture/skybox/posx.jpg",
		"asset/texture/skybox/negx.jpg",
		"asset/texture/skybox/posy.jpg",
		"asset/texture/skybox/negy.jpg",
		"asset/texture/skybox/posz.jpg",
		"asset/texture/skybox/negz.jpg"
	};
	//Read Image
	int w, h, n, num_faces = 6;
	stbi_loadf(faces[0].c_str(), &w, &h, &n, 4);
	size_t face_size = w * h * 4;
	float* h_data = new float[face_size * num_faces];
	for (int i = 0; i < num_faces; i++) {
		float* image = stbi_loadf(faces[i].c_str(), &w, &h, &n, 4);
		memcpy(h_data + face_size * i, image, face_size * sizeof(float));
		delete image;
	}

	//Cuda Malloc
	hipChannelFormatDesc format = hipCreateChannelDesc(32, 32, 32, 32, hipChannelFormatKindFloat);
	hipArray_t cu3dArr;
	gpuErrchk(hipMalloc3DArray(&cu3dArr, &format, make_hipExtent(w, h, num_faces), hipArrayCubemap));
	hipMemcpy3DParms myparms = { 0 };
	myparms.srcPos = make_hipPos(0, 0, 0);
	myparms.dstPos = make_hipPos(0, 0, 0);
	myparms.srcPtr = make_hipPitchedPtr(h_data, w * 4 * sizeof(float), w, h);
	myparms.dstArray = cu3dArr;
	myparms.extent = make_hipExtent(w, h, num_faces);
	myparms.kind = hipMemcpyHostToDevice;
	gpuErrchk(hipMemcpy3D(&myparms));

	//Init texture
	skyBox.addressMode[0] = hipAddressModeWrap;
	skyBox.addressMode[1] = hipAddressModeWrap;
	skyBox.addressMode[2] = hipAddressModeWrap;
	skyBox.filterMode = hipFilterModeLinear;
	skyBox.normalized = true;
	gpuErrchk(hipBindTextureToArray(&skyBox, cu3dArr, &format));

	delete h_data;
}

void initRayCasting()
{
	initCudaTexture();
	initSkyBox();
	//hipChannelFormatDesc format = hipCreateChannelDesc(32, 32, 32, 32, hipChannelFormatKindUnsigned);
	//size_t octree_size = size_t(h_curIdx) * sizeof(Node);
	//size_t offset = 0;
	//cudaStatus = hipBindTexture(&offset, &octree, d_node, &format, octree_size);
	//if (cudaStatus != hipSuccess) printf("hipBindTexture Failed, error: %s\n", hipGetErrorString(cudaStatus));
	gpuErrchk(hipDeviceSetLimit(hipLimitStackSize, 1024 * 16));

}






