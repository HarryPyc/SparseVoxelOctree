#include "hip/hip_runtime.h"
#include "Octree.cuh"
#include ""
#include <glm/gtc/integer.hpp>
#include <time.h>
#define NULLPTR 0U
#define MARKED 0xFFFFFFFF
extern VoxelizationInfo Info;
extern uint h_MIPMAP;
__constant__ VoxelizationInfo d_Info;
__constant__ uint maxLevel, curLevel, voxelCount;
__constant__ uint start, end, MIPMAP;
//__constant__ VoxelizationInfo d_Info;
__device__ uint curIdx;
texture<float4, 2, hipReadModeElementType> frontTex, backTex;

__host__ __device__ Node::Node() : voxel() {
	ptr = NULLPTR;
}

__host__ __device__ Node::~Node() {
}

__global__ void MarkKernel(Node* d_node, uint* d_idx) {
	const unsigned int x = blockDim.x * blockIdx.x + threadIdx.x;
	if (x >= voxelCount) return;
	uint nodeIdx = 0;
	//printf("marked: %u", MARKED);
	if (d_node[nodeIdx].ptr == NULLPTR || d_node[nodeIdx].ptr == MARKED) {
		d_node[nodeIdx].ptr = MARKED;
		return;
	}

	glm::uvec3 idx = ConvUintToUvec3(d_idx[x]), _idx = glm::uvec3(0);

	for (uint i = 0; i <= curLevel; i++) {
		_idx = idx % glm::uvec3(1 << (maxLevel - i)) / glm::uvec3(1 << (maxLevel - i - 1));
		nodeIdx = d_node[nodeIdx].ptr + _idx.x + _idx.y * 2 + _idx.z * 4;
		if (d_node[nodeIdx].ptr == NULLPTR || d_node[nodeIdx].ptr == MARKED) {
			d_node[nodeIdx].ptr = MARKED;
			return;
		}
	}
	
}
__global__ void AllocateKernel(Node* d_node) {
	const unsigned int x = blockDim.x * blockIdx.x + threadIdx.x + start;
	if (x >= end) return;

	if(d_node[x].ptr == MARKED)
		d_node[x].ptr = atomicAdd(&curIdx, 8);

}


__global__ void MemcpyVoxelToLeafNodeKernel(Node* d_node, Voxel* voxelSrc, uint* d_idx) {
	const unsigned int x = blockDim.x * blockIdx.x + threadIdx.x;
	if (x >= voxelCount) return;
	uint nodeIdx = 0;

	glm::uvec3 idx = ConvUintToUvec3(d_idx[x]), _idx = glm::uvec3(0);

	for (uint i = 0; i < maxLevel; i++) {
		_idx = idx % glm::uvec3(1 << (maxLevel - i)) / glm::uvec3(1 << (maxLevel - i - 1));
		//printf("[%i,%i,%i]\n", _idx.x, _idx.y, _idx.z);
		nodeIdx = d_node[nodeIdx].ptr + _idx.x + _idx.y * 2 + _idx.z * 4;
	}
	//printf("voxel[%u], node[%u]\n", x, nodeIdx);
	//leaf node pointer points to voxel list
	d_node[nodeIdx].voxel = voxelSrc[x];

}
__global__ void MimmapKernel(Node* d_node, Voxel* d_voxel) {
	const unsigned int x = blockDim.x * blockIdx.x + threadIdx.x + start;
	if (x >= end) return;
	const Node root = d_node[x];
	if (root.ptr == NULLPTR) return;

	glm::vec3 color(0.f), normal(0.f);
	float counter = 0.f;
	for (uint i = 0; i < 8; i++) {
		glm::vec3 c, n;
		Voxel voxel = d_node[root.ptr + i].voxel;
		if (!voxel.empty()) {
			voxel.GetInfo(c, n);
			color += c, normal += n;
			counter++;
		}
	}
	d_node[x].voxel.SetInfo(color/counter, normal/counter);
}

void initCudaTexture()
{
	frontTex.addressMode[0] = hipAddressModeWrap;
	frontTex.addressMode[1] = hipAddressModeWrap;
	frontTex.filterMode = hipFilterModeLinear;
	frontTex.normalized = true;

	backTex.addressMode[0] = hipAddressModeWrap;
	backTex.addressMode[1] = hipAddressModeWrap;
	backTex.filterMode = hipFilterModeLinear;
	backTex.normalized = true;
}

void OctreeConstruction(Node*& d_node, Voxel*& d_voxel, uint* d_idx)
{
	
	hipError_t cudaStatus;
	const uint h_maxLevel = glm::log2(Info.Dim);
	uint h_start = 0, h_curIdx = 1;
	uint* startArr = new uint[h_maxLevel], *endArr = new uint[h_maxLevel];
	hipMemcpyToSymbol(HIP_SYMBOL(maxLevel), &h_maxLevel, sizeof(uint));
	hipMemcpyToSymbol(HIP_SYMBOL(voxelCount), &Info.Counter, sizeof(uint));
	hipMemcpyToSymbol(HIP_SYMBOL(start), &h_start, sizeof(uint));
	hipMemcpyToSymbol(HIP_SYMBOL(end), &h_curIdx, sizeof(uint));
	hipMemcpyToSymbol(HIP_SYMBOL(curIdx), &h_curIdx, sizeof(uint));

	cudaStatus = hipMalloc((void**)&d_node, Info.Counter * sizeof(Node));
	if (cudaStatus != hipSuccess) printf("d_Node hipMalloc Failed\n");
	clock_t time = clock();
	for (uint i = 0; i < h_maxLevel; i++) {
		hipMemcpyToSymbol(HIP_SYMBOL(curLevel), &i, sizeof(uint));

		//Mark Node that need to be subdivied
		dim3 blockDim = 256, gridDim = Info.Counter / blockDim.x + 1;
		MarkKernel << <gridDim, blockDim >> > (d_node, d_idx);
		cudaStatus = hipGetLastError();
		if (cudaStatus != hipSuccess) printf("MarkKernel launch Failed, error: %s\n", hipGetErrorString(cudaStatus));
		cudaStatus = hipDeviceSynchronize();
		if (cudaStatus != hipSuccess) printf("MarkKernel hipDeviceSynchronize Failed\n");
		//Allocate new node from node pool
		gridDim = (h_curIdx - h_start) / blockDim.x + 1;
		AllocateKernel << <gridDim, blockDim >> > (d_node);
		if (cudaStatus != hipSuccess) printf("AllocateKernel launch Failed, error: %s\n", hipGetErrorString(cudaStatus));
		cudaStatus = hipDeviceSynchronize();
		if (cudaStatus != hipSuccess) printf("AllocateKernel hipDeviceSynchronize Failed\n");
		h_start = h_curIdx;
		hipMemcpyFromSymbol(&h_curIdx, HIP_SYMBOL(curIdx), sizeof(uint));
		hipMemcpyToSymbol(HIP_SYMBOL(start), &h_start, sizeof(uint));
		hipMemcpyToSymbol(HIP_SYMBOL(end), &h_curIdx, sizeof(uint));
		startArr[i] = h_start, endArr[i] = h_curIdx;
	}
	
	//Copy voxel to leaf node
	dim3 blockDim = 256, gridDim = Info.Counter / blockDim.x + 1;
	MemcpyVoxelToLeafNodeKernel << <gridDim, blockDim >> > (d_node, d_voxel, d_idx);
	if (cudaStatus != hipSuccess) printf("MemcpyVoxelToLeafNodeKernel launch Failed, error: %s\n", hipGetErrorString(cudaStatus));
	cudaStatus = hipDeviceSynchronize();
	if (cudaStatus != hipSuccess) printf("MemcpyVoxelToLeafNodeKernel hipDeviceSynchronize Failed\n");
	cudaStatus = hipFree(d_voxel);
	if (cudaStatus != hipSuccess) printf("d_voxel hipFree Failed\n");
	cudaStatus = hipFree(d_idx);
	if (cudaStatus != hipSuccess) printf("d_idx hipFree Failed\n");

	//Mimmap voxel value from bottom to up
	for (int i = h_maxLevel - 2; i >= 0; i--) {
		hipMemcpyToSymbol(HIP_SYMBOL(curLevel), &i, 4);
		hipMemcpyToSymbol(HIP_SYMBOL(start), startArr + i, 4);
		hipMemcpyToSymbol(HIP_SYMBOL(end), endArr + i, 4);

		gridDim = (endArr[i] - startArr[i]) / blockDim.x + 1;
		MimmapKernel << <gridDim, blockDim >> > (d_node, d_voxel);
		if (cudaStatus != hipSuccess) printf("MimmapKernel launch Failed, error: %s\n", hipGetErrorString(cudaStatus));
		cudaStatus = hipDeviceSynchronize();
		if (cudaStatus != hipSuccess) printf("MimmapKernel hipDeviceSynchronize Failed\n");
	}
	time = clock() - time;
	printf("Octree Constructed, time: %f\n", float(time) / CLOCKS_PER_SEC);
	printf("Octree Total Nodes : %u\n", h_curIdx);
	//Node h_node[8777];
	//hipMemcpy(h_node, d_node, sizeof(Node) * 8777, hipMemcpyDeviceToHost);
	delete[] startArr, delete[] endArr;
	initCudaTexture();
	cudaStatus = hipDeviceSetLimit(hipLimitStackSize, 1024 * 10);
	if (cudaStatus != hipSuccess) printf("hipDeviceSetLimit Failed\n");
}
struct Ray {
	glm::vec3 o, d, invD;
	__device__ Ray(glm::vec3 origin, glm::vec3 dir) : o(origin), d(dir) {
		invD = glm::vec3(1.f) / dir;
	}
	__device__ ~Ray() {};
	__device__ inline bool RayAABBIntersection(glm::vec3 minAABB, glm::vec3 maxAABB, float &t) {
		glm::vec3 t0s = (minAABB - o) * invD;
		glm::vec3 t1s = (maxAABB - o) * invD;

		glm::vec3 tsmaller = glm::min(t0s, t1s);
		glm::vec3 tbigger = glm::max(t0s, t1s);

		float tmin = glm::max(-999.f, glm::max(tsmaller[0], glm::max(tsmaller[1], tsmaller[2])));
		float tmax = glm::min(999.f, glm::min(tbigger[0], glm::min(tbigger[1], tbigger[2])));
		t = (tmin + tmax) / 2.f;
		return (tmin < tmax);
	}
};
struct HitInfo {
	glm::uvec3 idx;
	float t;
	__device__ HitInfo() {};
	__device__ HitInfo(glm::uvec3 _idx, float _t) : idx(_idx), t(_t) {};
};
__device__ Voxel OctreeTraverse(Node* d_node, Node root, Ray ray, glm::vec3 minAABB, uint currentLevel, float& t) {
	if (root.ptr == NULLPTR)
		return root.voxel;
	if (currentLevel == MIPMAP)
		return root.voxel;

	currentLevel++;
	HitInfo hits[8];
	int counter = 0;
	float temp = 999.f;
	Voxel res;
	const float delta = d_Info.delta / float((1 << (currentLevel)));
	for (int i = 0; i < 2; i++)
		for (int j = 0; j < 2; j++)
			for (int k = 0; k < 2; k++) {
				glm::uvec3 idx(i, j, k);
				glm::vec3 _minAABB = minAABB + glm::vec3(idx) * delta;
				float _t;
				if (ray.RayAABBIntersection(_minAABB, _minAABB + delta, _t)) {
					HitInfo hit(idx, _t);
					hits[counter++] = hit;
				}
			}
	for(int i = 0; i < counter - 1; i++)
		for (int j = 0; j < counter - i - 1; j++) {
			if (hits[j].t > hits[j + 1].t) {
				HitInfo temp = hits[j];
				hits[j] = hits[j + 1];
				hits[j + 1] = temp;
			}
		}
	for (int i = 0; i < counter; i++) {
		Node _root = d_node[root.ptr + hits[i].idx.x + hits[i].idx.y * 2 + hits[i].idx.z * 4];
		glm::vec3 _minAABB = minAABB + glm::vec3(hits[i].idx) * delta;
		Voxel voxel = OctreeTraverse(d_node, _root, ray, _minAABB, currentLevel,  t);
		if (!voxel.empty()) {
			if (currentLevel == MIPMAP)
				t = hits[i].t;
			return voxel;
		}
	}
	return res;
}
__global__ void RayCastKernel(uint* d_pbo, Node* d_node, const uint w, const uint h) {
	const unsigned int x = blockDim.x * blockIdx.x + threadIdx.x,
			y = blockDim.y * blockIdx.y + threadIdx.y;
	if (x >= w || y >= h) return;
	d_pbo[y * w + x] = 0;
	const float u = float(x) / float(w), v = float(y) / float(h);
	float4 frontSample = tex2D(frontTex, u, v), backSample = tex2D(backTex, u, v);
	if (frontSample.w < 1.f) return;
		
	glm::vec3 frontPos(frontSample.x, frontSample.y, frontSample.z),
		backPos(backSample.x, backSample.y, backSample.z);
	glm::vec3 dir = glm::normalize(backPos - frontPos);
	Ray ray(frontPos, dir);

	glm::vec3 color, pos;
	float t = 999.f;
	Voxel voxel = OctreeTraverse(d_node, d_node[0], ray, d_Info.minAABB, 0, t);
	pos = ray.o + t * ray.d;
	color = voxel.PhongLighting(pos);
	d_pbo[y * w + x] = ConvVec4ToUint(glm::vec4(color, 1));

}

void RayCastingOctree(uint* d_pbo, hipArray_t front, hipArray_t back, Node* d_node)
{
	if (hipMemcpyToSymbol(HIP_SYMBOL(d_Info), &Info, sizeof(VoxelizationInfo)) != hipSuccess)
		printf("hipMemcpy to constant failed\n");
	hipMemcpyToSymbol(HIP_SYMBOL(MIPMAP), &h_MIPMAP, 4);

	hipChannelFormatDesc format = hipCreateChannelDesc(32, 32, 32, 32, hipChannelFormatKindFloat);
	hipError_t cudaStatus;
	if (hipBindTextureToArray(&frontTex, front, &format) != hipSuccess)
		printf("front texture bind failed\n");
	if (hipBindTextureToArray(&backTex, back, &format) != hipSuccess)
		printf("back texture bind failed\n");
	//launch cuda kernel
	dim3 blockDim(16, 16, 1), gridDim(WINDOW_WIDTH / blockDim.x + 1, WINDOW_HEIGHT / blockDim.y + 1, 1);
	RayCastKernel << <gridDim, blockDim >> > (d_pbo, d_node, WINDOW_WIDTH, WINDOW_HEIGHT);
	cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess) printf("raymarching cuda Launch Kernel Failed\n");
	cudaStatus = hipDeviceSynchronize();
	if(cudaStatus != hipSuccess)
		printf("raymarching hipDeviceSynchronize Failed, error: %s\n", hipGetErrorString(cudaStatus));

	if (hipUnbindTexture(frontTex) != hipSuccess)
		printf("cuda unbind texture failed\n");
	if (hipUnbindTexture(backTex) != hipSuccess)
		printf("cuda unbind texture failed\n");
}



